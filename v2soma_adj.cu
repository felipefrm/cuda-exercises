
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4 // quantidades de numeros
#define I 2 // adjacentes

// codigo device
__global__ void soma_adj(int *a){
    int ind = threadIdx.x;
    int pos_inicio = ind - I;
    int pos_final = ind + I + 1;
    int soma = 0;
    if (ind < N){
      for (int i=pos_inicio; i<pos_final; i++){
        if (i >= 0 && i < N)
          soma += a[i];
      }
      __syncthreads();
      a[ind] = soma;
    }
}


// Código host
int main(){
  int a[N];
  int* dev_a;

  // Inicializando as variaveis do host
  for (int i = 0; i < N; i++)
    a[i] = i+1;

  // Alocando espaço para as variaveis da GPU
  hipMalloc((void**)&dev_a, N*sizeof(int));

  // Copiando as variaveis da CPU para a GPU
  hipMemcpy(dev_a, &a, N*sizeof(int), hipMemcpyHostToDevice);

  // Chamada à função da GPU (kernel)
  // A terceira dimensao é omitida, ficando implícito o valor 1.
  soma_adj<<<1, N>>>(dev_a);

  // Copiando o resultado da GPU para CPU
  hipMemcpy(&a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);

  // Visualizando o resultado
  for (int i=0; i<N; i++)
    printf("%d \n", a[i]);

  // Liberando a memoria na GPU
  hipFree(dev_a);

  return 0;
}
